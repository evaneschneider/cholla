
#include <hip/hip_runtime.h>
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"cooling_cuda.h"
#include"error_handling.h"
#include"io.h"



Real CTU_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt,int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  int  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
  #endif


  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // initial input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_L, *Q_R, *F;
  // array to hold zero values for H correction (necessary to pass to Roe solver)
  Real *etah;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;
  #if defined COOLING_GPU
  // array of timesteps for dt calculation (cooling restriction)
  Real *dev_dt_array;
  #endif  

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_L, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_R, n_fields*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F,   (n_fields)*n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah, n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
  #if defined COOLING_GPU
  CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
  #endif  

  // copy the conserved variable array onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();


  // Step 1: Do the reconstruction
  #ifdef PCM
  PCM_Reconstruction_1D<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMP
  PLMP_cuda<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMC
  PLMC_cuda<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMP
  PPMP_cuda<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMC
  PPMC_cuda<<<dimGrid,dimBlock>>>(dev_conserved, Q_L, Q_R, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif

  
  // Step 2: Calculate the fluxes
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, etah, 0, n_fields);
  #endif
  #ifdef HLLC 
  Calculate_HLLC_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_L, Q_R, F, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 3: Update the conserved variable array
  Update_Conserved_Variables_1D<<<dimGrid,dimBlock>>>(dev_conserved, F, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();
   

  // Sychronize the total and internal energy, if using dual-energy formalism
  #ifdef DE
  Sync_Energies_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif


  // Apply cooling
  #ifdef COOLING_GPU
  cooling_kernel<<<dimGrid,dimBlock>>>(dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gama, dev_dti_array);
  CudaCheckError();
  #endif

  // Calculate the next timestep
  Calc_dt_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(host_conserved1, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #if defined COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // find maximum inverse timestep from cooling time
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  if (min_dt < C_cfl/max_dti) {
    max_dti = C_cfl/min_dt;
  }
  #endif

  // free the CPU memory
  free(host_dti_array);
  #if defined COOLING_GPU
  free(host_dt_array);  
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_L);
  cudaFree(Q_R);
  cudaFree(F);
  cudaFree(etah);
  cudaFree(dev_dti_array);
  #if defined COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

  // return the maximum inverse timestep
  return max_dti;


}


#endif //CUDA
