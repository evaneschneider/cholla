
#include <hip/hip_runtime.h>
/*! \file hydro_cuda.cu
 *  \brief Definitions of functions used in all cuda integration algorithms. */
#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int x_off, int n_ghost, Real dx, Real xbound, Real dt, Real gamma, int n_fields)
{
  int id;
  #if defined(DE) || defined(STATIC_GRAV)
  Real d, d_inv, vx;  
  #endif
  #ifdef DE
  Real vx_imo, vx_ipo, vy, vz, P;
  #endif
  #ifdef STATIC_GRAV
  Real gx, d_n, d_inv_n, vx_n;
  gx = 0.0;
  #endif
  
  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    #if defined(DE) || defined(STATIC_GRAV)
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    #endif
    #ifdef DE
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    vx_imo = dev_conserved[1*n_cells + id-1]/dev_conserved[id-1];
    vx_ipo = dev_conserved[1*n_cells + id+1]/dev_conserved[id+1];
    #endif
  
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F[(5+i)*n_cells + id-1] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F[(n_fields-1)*n_cells + id-1] - dev_F[(n_fields-1)*n_cells + id])
                                  +  dtodx * P * 0.5 * (vx_imo - vx_ipo);
    #endif
    #ifdef STATIC_GRAV // add gravitational source terms, time averaged from n to n+1
    calc_g_1D(id, x_off, n_ghost, dx, xbound, &gx);    
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n);
    #endif    
    //printf("%3d %e %e %e\n", id-n_ghost, fz, gcorr, gx);
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update. %f\n", id, dev_conserved[id]);
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure after final update.\n", id);
    */
  }


}


__global__ void Update_Conserved_Variables_2D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  #if defined (DE) || defined(STATIC_GRAV)
  Real d, d_inv, vx, vy;
  #endif
  #ifdef DE
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz, P;
  int ipo, jpo;
  #endif

  #ifdef STATIC_GRAV
  Real gx, gy, d_n, d_inv_n, vx_n, vy_n;
  //gx = 0.0;
  //gy = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  imo = xid-1 + yid*nx;
  jmo = xid + (yid-1)*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    #if defined (DE) || defined (STATIC_GRAV)
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    #endif
    #ifdef DE
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    ipo = xid+1 + yid*nx;
    jpo = xid + (yid+1)*nx;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    #endif
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                        +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
//    if (xid == 4) {
//      printf("%3d %f %f %f\n", yid, dev_conserved[5*n_cells + id]), dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id]), dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
//    }
    #endif
    // add gravitational source terms, time averaged from n to n+1                                 
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                  +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo));
    //if (dev_conserved[5*n_cells + id] < 0.0) printf("%3d %3d Negative internal energy after final update.\n", xid, yid);
    #endif
    #ifdef STATIC_GRAV 
    // calculate the gravitational acceleration as a function of x & y position
    calc_g_2D(xid, yid, x_off, y_off, n_ghost, dx, dy, xbound, ybound, &gx, &gy);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n);
    //dev_conserved[4*n_cells + id] += 0.5*dt*gx*(d*vx + d_n*vx_n)
    //                              +  0.5*dt*gy*(d*vy + d_n*vy_n);
    #endif
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id]) {
      printf("%3d %3d Thread crashed in final update. %f %f %f\n", xid, yid, dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dev_conserved[id]);
    }   
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0)
      printf("%3d %3d Negative pressure after final update. %f %f %f %f\n", xid, yid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, P);    
    */
  }

}



__global__ void Update_Conserved_Variables_3D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, 
                                              Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt,
                                              Real gamma, int n_fields)
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  #if defined (DE) || defined(STATIC_GRAV)
  Real d, d_inv, vx, vy, vz;
  #endif
  #ifdef DE
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo, P;
  int ipo, jpo, kpo;
  #endif

  #ifdef STATIC_GRAV
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  imo = xid-1 + yid*nx + zid*nx*ny;
  jmo = xid + (yid-1)*nx + zid*nx*ny;
  kmo = xid + yid*nx + (zid-1)*nx*ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    #if defined (DE) || defined(STATIC_GRAV)
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    #endif
    #ifdef DE
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before final update.\n");
    //if (P < 0.0) printf("%d Negative pressure before final update.\n", id);
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo]; 
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo]; 
    #endif

    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                  +  dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                  +  dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                  +  dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                  +  dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    //fz = dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
    //                              +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
    //                              +  dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                  +  dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                    +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                    +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }                              
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                  +  dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                                  +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
    #endif
    // add a density floor of n = 1.0e-3
    //dev_conserved[id] = fmax(dev_conserved[id], 0.6*MP*1.0e-3/DENSITY_UNIT);
    #ifdef STATIC_GRAV 
    calc_g_3D_CUDA(xid, yid, zid, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, &gx, &gy, &gz);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n)
                                  +  0.25*dt*gz*(d + d_n)*(vz + vz_n);
    #endif    
    
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id] || dev_conserved[4*n_cells + id] < 0.0 || dev_conserved[4*n_cells+id] != dev_conserved[4*n_cells+id]) {
      printf("%3d %3d %3d Thread crashed in final update. %e %e %e %e %e\n", xid+x_off, yid+y_off, zid+z_off, dev_conserved[id], dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dtodz*(dev_F_z[kmo]-dev_F_z[id]), dev_conserved[4*n_cells+id]);
      //printf("%3d %3d %3d Thread crashed in final update. %e %e\n", xid+x_off, yid+y_off, zid+z_off, dev_conserved[id], dev_conserved[4*n_cells+id]);
      //Real ge = dev_conserved[5*n_cells + id];
      //Real T = ge * (gamma-1.0)*SP_ENERGY_UNIT*0.6*MP/(d_n*KB);
      //printf("Internal energy: %e  Temperature: %e\n", ge, T);
    }
    
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%3d %3d %3d Negative pressure after final update. %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, 0.5*d*vz*vz, P);
    */
  }

}


__global__ void Sync_Energies_1D(Real *dev_conserved, int n_cells, int n_ghost, Real gamma, int n_fields)
{
  int id;
  Real d, d_inv, vx, vy, vz, P, E;
  Real ge1, ge2, Emax;
  int im1, ip1;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  
  im1 = max(id-1, n_ghost);
  ip1 = min(id+1, n_cells-n_ghost-1);

  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    // separately tracked internal energy 
    ge1 = dev_conserved[(n_fields-1)*n_cells + id];
    // internal energy calculated from total energy
    ge2 = dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz);
    // if the ratio of conservatively calculated internal energy to total energy
    // is greater than 1/1000, use the conservatively calculated internal energy
    // to do the internal energy update
    if (ge2/E > 0.001) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
      ge1 = ge2;
    }     
    // find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + im1], E);
    Emax = fmax(dev_conserved[4*n_cells + ip1], Emax);
    // if the ratio of conservatively calculated internal energy to max nearby total energy
    // is greater than 1/10, continue to use the conservatively calculated internal energy 
    if (ge2/Emax > 0.1) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
    }
    // sync the total energy with the internal energy 
    else {
      dev_conserved[4*n_cells + id] += ge1 - ge2;
    }
    /*
    // if the conservatively calculated internal energy is greater than the estimate of the truncation error,
    // use the internal energy computed from the total energy to do the update
    //find the max nearby velocity difference (estimate of truncation error) 
    vmax = fmax(fabs(vx-dev_conserved[1*n_cells + im1]/dev_conserved[im1]), fabs(dev_conserved[1*n_cells + ip1]/dev_conserved[ip1]-vx));
    //printf("%3d %f %f %f %f\n", id, ge1, ge2, vmax, 0.25*d*vmax*vmax);
    if (ge2 > 0.25*d*vmax*vmax) {
      dev_conserved[5*n_cells + id] = ge2;
      ge1 = ge2;
    }
    //else printf("%d Using ge1 %f %f %f %f\n", id, ge1, ge2, vmax, 0.25*d*vmax*vmax);
    */
    // update the total energy
     
    // recalculate the pressure 
    P = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);    
    if (P < 0.0) printf("%d Negative pressure after internal energy sync. %f %f \n", id, ge1, ge2);    
  }

}


__global__ void Sync_Energies_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, P, E;
  Real ge1, ge2, Emax;
  int imo, ipo, jmo, jpo;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;

  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    // separately tracked internal energy 
    ge1 =  dev_conserved[(n_fields-1)*n_cells + id];
    // internal energy calculated from total energy
    ge2 = dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz);
    // if the ratio of conservatively calculated internal energy to total energy
    // is greater than 1/1000, use the conservatively calculated internal energy
    // to do the internal energy update
    if (ge2/E > 0.001) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
      ge1 = ge2;
    }     
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
    // if the ratio of conservatively calculated internal energy to max nearby total energy
    // is greater than 1/10, continue to use the conservatively calculated internal energy 
    if (ge2/Emax > 0.1) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
    }
    // sync the total energy with the internal energy 
    else {
      dev_conserved[4*n_cells + id] += ge1 - ge2;
    }
    // recalculate the pressure 
    P = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);    
    if (P < 0.0) printf("%d Negative pressure after internal energy sync. %f %f \n", id, ge1, ge2);    
  }
}




__global__ void Sync_Energies_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E;
  Real ge1, ge2, Emax;
  int imo, ipo, jmo, jpo, kmo, kpo;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx + zid*nx*ny;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx + zid*nx*ny;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx + zid*nx*ny;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx + zid*nx*ny;
  kmo = max(zid-1, n_ghost);
  kmo = xid + yid*nx + kmo*nx*ny;
  kpo = min(zid+1, nz-n_ghost-1);
  kpo = xid + yid*nx + kpo*nx*ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    // don't do the energy sync if this thread has crashed
    if (E < 0.0 || E != E) return;
    // separately tracked internal energy 
    ge1 =  dev_conserved[(n_fields-1)*n_cells + id];
    // internal energy calculated from total energy
    ge2 = dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz);
    /*
    if (ge2 < 0.0) {
      Real T = (ge1/d) * (gamma-1.0)*SP_ENERGY_UNIT*0.6*MP/KB;
      printf("%3d %3d %3d Temperature: %e\n", xid, yid, zid, T);
    }
    */
    // if the ratio of conservatively calculated internal energy to total energy
    // is greater than 1/1000, use the conservatively calculated internal energy
    // to do the internal energy update
    if (ge2 > 0.0 && E > 0.0 && ge2/E > 0.001) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
      ge1 = ge2;
    }
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kpo]);
    // if the ratio of conservatively calculated internal energy to max nearby total energy
    // is greater than 1/10, continue to use the conservatively calculated internal energy 
    if (ge2/Emax > 0.1 && ge2 > 0.0 && Emax > 0.0) {
      dev_conserved[(n_fields-1)*n_cells + id] = ge2;
    }
    // sync the total energy with the internal energy 
    else {
      if (ge1 > 0.0) dev_conserved[4*n_cells + id] += ge1 - ge2;
      else dev_conserved[(n_fields-1)*n_cells+id] = ge2;
    }
    // recalculate the pressure 
    //Real P = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (P < 0.0) printf("%3d %3d %3d Negative pressure after internal energy sync. %f %f %f\n", xid, yid, zid, P/(gamma-1.0), ge1, ge2);    
  }
}



__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) TINY_NUMBER);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = (fabs(vx)+cs)/dx;
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];


}



__global__ void Calc_dt_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real dx, Real dy, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid, xid, yid, n_cells;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) 1.0e-20);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockId] = max_dti[0];

}


__global__ void Calc_dt_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, E, P, cs;
  int id, xid, yid, zid, n_cells;
  int tid;

  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  // and a thread id within the block  
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  = dev_conserved[4*n_cells + id];
    P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
    max_dti[tid] = fmax(max_dti[tid], (fabs(vz)+cs)/dz);
    max_dti[tid] = fmax(max_dti[tid], 0);
    Real n = d*DENSITY_UNIT / (1.27*MP);
    Real T = P*PRESSURE_UNIT/(n*KB);
    // if this thread has crashed or will be reset, don't include it in the timestep calculation
    if (d < 0 || d != d || P < 0 || P != P || E < 0 || E != E) max_dti[tid] = 0;
    //max_dti[tid] = (fabs(vx)+cs)/dx + (fabs(vy)+cs)/dy + (fabs(vz)+cs)/dz;
    /*
    if (0.3/max_dti[tid] < 1.0) {
      Real n = d*DENSITY_UNIT/(0.6*MP);
      Real T = P*PRESSURE_UNIT/(n*KB);
      printf("%3d %3d %3d n: %e  T: %e\n", xid, yid, zid, n, T);
    }
    */
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];

}


__device__ void calc_g_1D(int xid, int x_off, int n_ghost, Real dx, Real xbound, Real *gx)
{
  Real x_pos, r_disk, r_halo;
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;

  // for disk components, calculate polar r
  //r_disk = 0.220970869121;
  //r_disk = 6.85009694274;
  r_disk = 13.9211647546;
  //r_disk = 20.9922325665;
  // for halo, calculate spherical r
  r_halo = sqrt(x_pos*x_pos + r_disk*r_disk);

  // set properties of halo and disk (these must match initial conditions)
  Real a_disk_z, a_halo, M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20.0; // halo concentration
  R_h = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  z_d = 3.5/5.0; // disk scale height in kpc
  phi_0_h = GN * M_h / (log(1.0+c_vir) - c_vir / (1.0+c_vir));
  x = r_halo / R_h;
  
  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo = - phi_0_h * (log(1+x) - x/(1+x)) / (r_halo*r_halo);
  a_disk_z = - GN * M_d * x_pos * (R_d + sqrt(x_pos*x_pos + z_d*z_d)) / ( pow(r_disk*r_disk + pow(R_d + sqrt(x_pos*x_pos + z_d*z_d), 2), 1.5) * sqrt(x_pos*x_pos + z_d*z_d) );

  // total acceleration is the sum of the halo + disk components
  *gx = (x_pos/r_halo)*a_halo + a_disk_z;

  return;

}


__device__ void calc_g_2D(int xid, int yid, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real *gx, Real *gy)
{
  Real x_pos, y_pos, r, phi;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;

  // for Gresho, also need r & phi
  r = sqrt(x_pos*x_pos + y_pos*y_pos);
  phi = atan2(y_pos, x_pos);

/*
  // set acceleration to balance v_phi in Gresho problem
  if (r < 0.2) {
    *gx = -cos(phi)*25.0*r;
    *gy = -sin(phi)*25.0*r;
  }
  else if (r >= 0.2 && r < 0.4) {
    *gx = -cos(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
    *gy = -sin(phi)*(4.0 - 20.0*r + 25.0*r*r)/r;
  }
  else {
    *gx = 0.0;
    *gy = 0.0;
  }
*/
/*
  // set gravitational acceleration for Keplarian potential
  Real M;
  M = 1*Msun;
  *gx = -cos(phi)*GN*M/(r*r);
  *gy = -sin(phi)*GN*M/(r*r);
*/
  // set gravitational acceleration for Kuzmin disk + NFW halo
  Real a_d, a_h, a, M_vir, M_d, R_vir, R_d, R_s, M_h, c_vir, x;
  M_vir = 1.0e12; // viral mass of MW in M_sun
  M_d = 6.5e10; // mass of disk in M_sun (assume all gas)
  M_h = M_vir - M_d; // halo mass in M_sun
  R_vir = 261; // viral radius in kpc
  c_vir = 20; // halo concentration
  R_s = R_vir / c_vir; // halo scale length in kpc
  R_d = 3.5; // disk scale length in kpc
  
  // calculate acceleration
  x = r / R_s;
  a_d = GN * M_d * r * pow(r*r + R_d*R_d, -1.5);
  a_h = GN * M_h * (log(1+x)- x / (1+x)) / ((log(1+c_vir) - c_vir / (1+c_vir)) * r*r);
  a = a_d + a_h;

  *gx = -cos(phi)*a;
  *gy = -sin(phi)*a;

  return;
}


__device__ void calc_g_3D_CUDA(int xid, int yid, int zid, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real *gx, Real *gy, Real *gz)
{
  Real x_pos, y_pos, z_pos, r_disk, r_halo;
  // use the subgrid offset and global boundaries to calculate absolute positions on the grid
  x_pos = (x_off + xid - n_ghost + 0.5)*dx + xbound;
  y_pos = (y_off + yid - n_ghost + 0.5)*dy + ybound;
  z_pos = (z_off + zid - n_ghost + 0.5)*dz + zbound;

  // for disk components, calculate polar r
  r_disk = sqrt(x_pos*x_pos + y_pos*y_pos);
  // for halo, calculate spherical r
  r_halo = sqrt(x_pos*x_pos + y_pos*y_pos + z_pos*z_pos);

  // set properties of halo and disk (these must match initial conditions)
  Real a_disk_r, a_disk_z, a_halo, a_halo_r, a_halo_z;
  Real M_vir, M_d, R_vir, R_d, z_d, R_h, M_h, c_vir, phi_0_h, x;
  // MW model
  M_vir = 1.0e12; // viral mass of in M_sun
  M_d = 6.5e10; // viral mass of in M_sun
  R_d = 3.5; // disk scale length in kpc
  z_d = 3.5/5.0; // disk scale height in kpc
  R_vir = 261.; // virial radius in kpc
  c_vir = 20.0; // halo concentration
  // M82 model
  //M_vir = 5.0e10; // viral mass of in M_sun
  //M_d = 1.0e10; // mass of disk in M_sun
  //R_d = 0.8; // disk scale length in kpc
  //z_d = 0.15; // disk scale height in kpc
  //R_vir = R_d/0.015; // viral radius in kpc
  //c_vir = 10.0; // halo concentration

  M_h = M_vir - M_d; // halo mass in M_sun
  R_h = R_vir / c_vir; // halo scale length in kpc
  phi_0_h = GN * M_h / (log(1.0+c_vir) - c_vir / (1.0+c_vir));
  x = r_halo / R_h;
  
  // calculate acceleration due to NFW halo & Miyamoto-Nagai disk
  a_halo = - phi_0_h * (log(1+x) - x/(1+x)) / (r_halo*r_halo);
  a_halo_r = a_halo*(r_disk/r_halo);
  a_halo_z = a_halo*(z_pos/r_halo);
  a_disk_r = - GN * M_d * r_disk * pow(r_disk*r_disk+ pow(R_d + sqrt(z_pos*z_pos + z_d*z_d),2), -1.5);
  a_disk_z = - GN * M_d * z_pos * (R_d + sqrt(z_pos*z_pos + z_d*z_d)) / ( pow(r_disk*r_disk + pow(R_d + sqrt(z_pos*z_pos + z_d*z_d), 2), 1.5) * sqrt(z_pos*z_pos + z_d*z_d) );

  // total acceleration is the sum of the halo + disk components
  //*gx = 0.0;
  //*gy = 0.0;
  //*gz = 0.0;
  *gx = (x_pos/r_disk)*(a_disk_r+a_halo_r);
  *gy = (y_pos/r_disk)*(a_disk_r+a_halo_r);
  //*gx = x_pos*a_halo/r_halo + x_pos*a_disk/r_disk;
  //*gy = y_pos*a_halo/r_halo + y_pos*a_disk/r_disk;
  *gz = a_disk_z+a_halo_z;
  //*gz = (z_pos/r_halo)*a_halo;
  //*gz = (z_pos/r_halo)*a_halo + a_disk_z;

  return;
}



#endif //CUDA
